#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <hiprand/hiprand_kernel.h>

#include "main.h"

#define N_DIMENSION 2 //for the sake of clarity in the code

// Initialize state for random numbers 
__global__ void init_kernel(hiprandState *state, long seed) 
{
    int idx = threadIdx.x + blockIdx.x * blockDim.x; 
    hiprand_init(seed, idx, 0, state);
}

__device__ float d_evaluate(Position pos)
{
    #if defined(EGGHOLDER)
        // Eggholder function [-512, 404.2319] -959.6407
        return (-(pos.y + 47)) * sin(sqrt(abs(pos.y + (pos.x / 2) + 47))) - (pos.x * sin(sqrt(abs(pos.x - (pos.y + 47)))));
    #elif defined(EASOM)
        // Easom function [pi, pi] -1
        return -cos(pos.x) * cos(pos.y) * exp(-(pow(pos.x - M_PI, 2) + pow(pos.y - M_PI, 2)));
    #endif
}

__global__ void pso_kernel(Particle* swarm, Position* gbest, float &gbest_fitness, float inertia, float cognitive, float social, int n_particles, hiprandState* state)
{
    int id = threadIdx.x + blockIdx.x * blockDim.x;
    
    if(id < n_particles)
    {
        float r1 = hiprand_uniform(state);
        float r2 = hiprand_uniform(state);

        //update position and velocity
        swarm[id].velocity.x = swarm[id].velocity.x * inertia + cognitive * r1 * (swarm[id].pbest.x - swarm[id].position.x) + social * r2 * (gbest->x - swarm[id].position.x);
        swarm[id].velocity.y = swarm[id].velocity.y * inertia + cognitive * r1 * (swarm[id].pbest.y - swarm[id].position.y) + social * r2 * (gbest->y - swarm[id].position.y);

        //check constraints
        if (swarm[id].velocity.x > MAX_V)
            swarm[id].velocity.x = MAX_V;
        else if (swarm[id].velocity.x < MIN_V)
            swarm[id].velocity.x = MIN_V;
        
        if (swarm[id].velocity.y > MAX_V)
            swarm[id].velocity.y = MAX_V;
        else if (swarm[id].velocity.y < MIN_V)
            swarm[id].velocity.y = MIN_V;

        //update position
        swarm[id].position.x += swarm[id].velocity.x;
        swarm[id].position.y += swarm[id].velocity.y;

        //check constraints
        if (swarm[id].position.x > MAX_D)
            swarm[id].position.x = MAX_D;
        else if (swarm[id].position.x < MIN_D)
            swarm[id].position.x = MIN_D;
        
        if (swarm[id].position.y > MAX_D)
            swarm[id].position.y = MAX_D;
        else if (swarm[id].position.y < MIN_D)
            swarm[id].position.y = MIN_D;

        //update fitness
        float fitness = d_evaluate(swarm[id].position);

        //update pbest
        if (fitness < swarm[id].pbest_fitness)
        {
            swarm[id].pbest_fitness = fitness;
            swarm[id].pbest.x = swarm[id].position.x;
            swarm[id].pbest.y = swarm[id].position.y;
        }

        if (swarm[id].pbest_fitness < gbest_fitness)
        {
            gbest_fitness = swarm[id].pbest_fitness;
            gbest->x = swarm[id].position.x;
            gbest->y = swarm[id].position.y;
        }
    }
}

int main(int argc, char** argv){
    if(argc != 1)
    {
        n_iterations = atoi(argv[1]);
        n_particles = atoi(argv[2]);
    }

    //std::cout << "Iterations: " << n_iterations << "\tParticles: " << n_particles << std::endl;

    // Thread and block dimensions
    int threadsNum = 1024;
    int blocksNum = (n_particles + threadsNum - 1) / threadsNum;

    hiprandState* state;
    hipMalloc(&state, n_particles * sizeof(hiprandState));
    init_kernel<<<1, 1>>>(state, time(NULL));

    // initialize the particles 
    Particle *particles = new Particle[n_particles];

    // define dev pointers to particles and gbest gbest_fitness
    Particle *d_particles;
    float *d_gbest_fitness;
    Position *d_gbest;

    // meausre time of execution of the algorithm
    auto start = std::chrono::high_resolution_clock::now();

    hipMalloc((void **)&d_particles, n_particles * sizeof(Particle));
    hipMalloc((void **)&d_gbest_fitness, sizeof(float));
    hipMalloc((void **)&d_gbest, sizeof(Position));

    // Copy particles to device
    hipMemcpy(d_particles, particles, n_particles * sizeof(Particle), hipMemcpyHostToDevice);
    hipMemcpy(d_gbest_fitness, &gbest_fitness, sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_gbest, &gbest, sizeof(Position), hipMemcpyHostToDevice);

    // Launch the kernel for each iteration
    for(int it = 0; it < n_iterations; it++)
    {
        // adaptive parameter 
        inertia = (0.4f / (n_iterations * n_iterations)) * ((it - n_iterations) * (it - n_iterations))  + 0.4f;
        cognitive = (-3.f * ((float) it / (float) n_iterations)) + 3.5f;
        social = (3.f * ((float) it / (float) n_iterations)) + 0.5f;
        
        pso_kernel<<<blocksNum, threadsNum>>>(d_particles, d_gbest, *d_gbest_fitness, inertia, cognitive, social, n_particles, state);
    }

    // Copy gbest fitness and gbest to host
    hipMemcpy(&gbest_fitness, d_gbest_fitness, sizeof(float), hipMemcpyDeviceToHost);
    hipMemcpy(&gbest, d_gbest, N_DIMENSION * sizeof(float), hipMemcpyDeviceToHost);

    auto stop = std::chrono::high_resolution_clock::now();
    // duration as float
    std::chrono::duration<float, std::milli> duration = stop - start;
    
    // print the best position and fitness
    #ifdef CSV
        printf("CUDA;%.2f;%.2f;%.2f;%.2f\n", gbest.x, gbest.y, gbest_fitness, duration.count());
        //std::cout << "CUDA;" << gbest.x << ";" << gbest.y << ";" << gbest_fitness << ";" << duration.count() << std::endl;
    #else
        std::cout << "Best position: (" << gbest.x << " , " << gbest.y << ") | Best fitness: " << gbest_fitness << " | Time: " << duration.count() << " ms" << std::endl;
    #endif

    // Free memory
    hipFree(d_particles);
    hipFree(d_gbest_fitness);
    hipFree(d_gbest);
    hipFree(state);

    delete[] particles;

    return 0;
}
